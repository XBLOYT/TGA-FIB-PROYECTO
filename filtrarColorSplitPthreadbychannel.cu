#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/times.h>
#include <sys/resource.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

char *fileIN, *fileOUT;
unsigned char *image, *equalizedImage;
int width, height, pixelWidth; //meta info de la imagen
int histogram[768];
unsigned int PREC = 10000;

__global__ void HistoK(unsigned int N, unsigned char *image, int *h){
	__shared__ int h_private[768];
	int i = (blockIdx.x*blockDim.x + threadIdx.x);
	int stride = blockDim.x * gridDim.x;
	int j = threadIdx.x;
	int k = j;
	while(j < 768){
		h_private[j] = 0;
		j += 256;
	}
	__syncthreads();
	while (i < N) {
		unsigned char color = image[i];
		atomicAdd(&h_private[color + 256*(i%3)], 1);
		i = i + stride;
	}
	__syncthreads();
	while(k < 768){
		atomicAdd(&h[k], h_private[k]);
		k += 256;
	}
}

__global__ void Equalize(unsigned int N, unsigned char *image, int *minmaxArray, unsigned int *prob){
	__shared__ unsigned int PREC;
	int i = (blockIdx.x*blockDim.x + threadIdx.x);
	int stride = blockDim.x * gridDim.x;
	if(i == 0){
		PREC = 10000;
	}
	while (i < N){
		unsigned char color = (prob[image[i] + 256*(i%3)] * minmaxArray[2 + 3*(i%3)] + PREC * minmaxArray[0 + 3*(i%3)])/PREC;
		image[i] = color;
		i += stride;
	}
}

void CheckCudaError(char sms[], int line){
	hipError_t error;

	error = hipGetLastError();
	if(error){
		printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
		exit(EXIT_FAILURE);
	}
}

float GetTime(void){
	struct timeval tim;
	struct rusage ru;
	getrusage(RUSAGE_SELF, &ru);
	tim = ru.ru_utime;
	return ((double) tim.tv_sec + (double)tim.tv_usec / 1000000.0)*1000.0;
}

int minHisto(int i_histogram[], int l, int r){
	for(int i = l; i <= r; i++){
		if(i_histogram[i] != 0) return i;
	}
	return -1;
}

int maxHisto(int i_histogram[], int l, int r){
	for(int i = r; i >= l; i--){
		if(i_histogram[i] != 0) return i;
	}
	return -1;
}

int main(int argc, char** argv)
{
  unsigned int N;
  unsigned int numBytesImage, numBytesHisto, numBytesMinMax;
  unsigned int nBlocks, nThreads;
  int *d_histogram, *d_minmaxArray;
  unsigned char *d_image;
  unsigned int *d_prob;
  float t1, t2, tiempoKHistoK, tiempoKEqualize, tiempoProcessing, tiempoHtoD, tiempoDtoH, tiempoKernels;

  hipEvent_t E1, E2, E3, E4, E5, E6;  

  // Ficheros de entrada y de salida 
  if (argc == 3) { fileIN = argv[1]; fileOUT = argv[2]; }
  else { printf("Usage: ./exe fileIN fileOUT (equalize)\n"); exit(0); }

  t1 = GetTime();
  printf("Reading image...\n");
  image = stbi_load(fileIN, &width, &height, &pixelWidth, 0);
  if (!image) {
    fprintf(stderr, "Couldn't load image.\n");
     return (-1);
  }
  printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);

  for (int i = 0; i < 768; i++) {
	  histogram[i] = 0;
  }
  printf("Filtrando\n");
  //Calculamos histograma:

  N = width * height * 3;
  nThreads = 256;
  nBlocks = (N + nThreads-1)/nThreads;

  numBytesImage = N * sizeof(unsigned char);
  numBytesHisto = 768 * sizeof(int);
  numBytesMinMax = 9 * sizeof(int);

  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);
  hipEventCreate(&E4);
  hipEventCreate(&E5);
  hipEventCreate(&E6);

  hipEventRecord(E5, 0);
  hipEventSynchronize(E5);
  hipMalloc((void**)&d_image, numBytesImage);
  hipMalloc((void**)&d_histogram, numBytesHisto);
  hipMalloc((void**)&d_minmaxArray, numBytesMinMax);
  hipMalloc((void**)&d_prob, numBytesHisto);
  CheckCudaError((char*) "Error de Malloc ", __LINE__);
 
  hipMemcpy(d_image, image, numBytesImage, hipMemcpyHostToDevice);
  hipMemcpy(d_histogram, histogram, numBytesHisto, hipMemcpyHostToDevice);
  CheckCudaError((char*) "Error de Memcpy HtoD", __LINE__);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);
  HistoK<<<nBlocks, nThreads>>>(N, d_image, d_histogram);
  CheckCudaError((char*) "Error de HistoK", __LINE__);
  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);
  
  hipMemcpy(histogram, d_histogram, numBytesHisto, hipMemcpyDeviceToHost);
  CheckCudaError((char*) "Error de Memcpy DtoH", __LINE__);

  int minR = minHisto(histogram, 0, 255), minG = minHisto(histogram, 256, 511), minB = minHisto(histogram, 512, 767);
  int maxR = maxHisto(histogram, 0, 255), maxG = maxHisto(histogram, 256, 511), maxB = maxHisto(histogram, 512, 767);
  int maxminR = maxR - minR, maxminG = maxG - minG, maxminB = maxB - minB;
  unsigned int prob[768];
  int length = 256;
  prob[0] = PREC * histogram[0]/(width*height);
  prob[256] = PREC * histogram[256]/(width*height);
  prob[512] = PREC * histogram[512]/(width*height);
  for(int i = 1; i < length; ++i){
	prob[i] = prob[i-1] + PREC * histogram[i]/(width*height);
	prob[i + 256] = prob[i + 256 -1] + PREC * histogram[i + 256]/(width*height);
	prob[i + 512] = prob[i + 512 -1] + PREC * histogram[i + 512]/(width*height);
  }
  int auxMinmaxArray[9] = {minR, maxR, maxminR, minG, maxG, maxminG, minB, maxB, maxminB};

  hipMemcpy(d_minmaxArray, auxMinmaxArray, numBytesMinMax, hipMemcpyHostToDevice);
  hipMemcpy(d_prob, prob, numBytesHisto, hipMemcpyHostToDevice);
  CheckCudaError((char*) "Error de Memcpy HtoD", __LINE__);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);
  Equalize<<<nBlocks, nThreads>>>(N, d_image, d_minmaxArray, d_prob);
  CheckCudaError((char*) "Error de Equalize", __LINE__);
  hipEventRecord(E4, 0);
  hipEventSynchronize(E4);

  hipMemcpy(image, d_image, numBytesImage, hipMemcpyDeviceToHost);
  CheckCudaError((char*) "Error de Memcpy DtoH ", __LINE__);

  hipFree(d_image);
  hipFree(d_histogram);
  hipFree(d_minmaxArray);
  hipFree(d_prob);
  CheckCudaError((char*) "Error de Free ", __LINE__);
  hipEventRecord(E6, 0);
  hipEventSynchronize(E6);

  hipEventElapsedTime(&tiempoKHistoK, E1, E2);
  hipEventElapsedTime(&tiempoKEqualize, E3, E4);
  hipEventElapsedTime(&tiempoProcessing, E5, E6);
  hipEventElapsedTime(&tiempoHtoD, E5, E1);
  hipEventElapsedTime(&tiempoDtoH, E4, E6);
  hipEventElapsedTime(&tiempoKernels, E1, E4);

  hipEventDestroy(E1);
  hipEventDestroy(E2);
  hipEventDestroy(E3);
  hipEventDestroy(E4);
  hipEventDestroy(E5);
  hipEventDestroy(E6);

  printf("Escribiendo\n");
  //ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT,width,height,pixelWidth,image,0);
  t2 = GetTime();
  printf("---FILTRAR ColorSplitP---\n");
  printf("tiempo Global: %4.6f milseg\n", t2-t1);
  printf("tiempo HtoD: %4.6f milseg\n", tiempoHtoD);
  printf("tiempo DtoH: %4.6f milseg\n", tiempoDtoH);
  printf("tiempo Kernel HistoK: %4.6f milseg\n", tiempoKHistoK);
  printf("tiempo Kernel Equalize: %4.6f milseg\n", tiempoKEqualize);
  printf("tiempo Kernels juntos: %4.6f milseg\n", tiempoKernels);
  printf("tiempo Ecualización: %4.6f milseg\n", tiempoProcessing);
  printf("Ancho de banda: %4.2f MB/s\n", 0.000001*((width*height*3)/(tiempoProcessing*0.001)));
}

