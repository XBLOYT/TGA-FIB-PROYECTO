#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/times.h>
#include <sys/resource.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

char *fileIN, *fileOUT;
unsigned char *image, *equalizedImage;
int width, height, pixelWidth; //meta info de la imagen
int histogramR[256], histogramB[256], histogramG[256];
unsigned int PREC = 10000;

__global__ void HistoR(unsigned int N, unsigned char *image, int *h){
	__shared__ int h_private[256];
	int i = 3*(blockIdx.x*blockDim.x + threadIdx.x);
	int stride = blockDim.x * gridDim.x;
	int j = threadIdx.x;
	h_private[j] = 0;
	__syncthreads();
	while (i < N) {
		unsigned char color = image[i];
		atomicAdd(&h_private[color], 1);
		i = i + stride;
	}
	__syncthreads();
	atomicAdd(&h[j], h_private[j]);
}

__global__ void HistoG(unsigned int N, unsigned char *image, int *h){
	__shared__ int h_private[256];
	int i = 3*(blockIdx.x*blockDim.x + threadIdx.x)+1;
	int stride = blockDim.x * gridDim.x;
	int j = threadIdx.x;
	h_private[j] = 0;
	__syncthreads();
	while (i < N) {
		unsigned char color = image[i];
		atomicAdd(&h_private[color], 1);
		i = i + stride;
	}
	__syncthreads();
	atomicAdd(&h[j], h_private[j]);
}

__global__ void HistoB(unsigned int N, unsigned char *image, int *h){
	__shared__ int h_private[256];
	int i = 3*(blockIdx.x*blockDim.x + threadIdx.x)+2;
	int stride = blockDim.x * gridDim.x;
	int j = threadIdx.x;
	h_private[j] = 0;
	__syncthreads();
	while (i < N) {
		unsigned char color = image[i];
		atomicAdd(&h_private[color], 1);
		i = i + stride;
	}
	__syncthreads();
	atomicAdd(&h[j], h_private[j]);
}

__global__ void EqualizeR(unsigned int N, unsigned char *image, int *minmaxArray, unsigned int *prob){
	__shared__ unsigned int PREC;
	int i = 3*(blockIdx.x*blockDim.x + threadIdx.x);
	int stride = blockDim.x * gridDim.x;
	PREC = 10000;
	while (i < N){
		unsigned char color = (prob[image[i]] * minmaxArray[2] + PREC * minmaxArray[0])/PREC;
		image[i] = color;
		i += stride;
	}
}

__global__ void EqualizeG(unsigned int N, unsigned char *image, int *minmaxArray, unsigned int *prob){
	__shared__ unsigned int PREC;
	int i = 3*(blockIdx.x*blockDim.x + threadIdx.x);
	int stride = blockDim.x * gridDim.x;
	PREC = 10000;
	while (i < N){
		unsigned char color = (prob[image[i+1]] * minmaxArray[5] + PREC * minmaxArray[3])/PREC;
		image[i+1] = color;
		i += stride;
	}
}

__global__ void EqualizeB(unsigned int N, unsigned char *image, int *minmaxArray, unsigned int *prob){
	__shared__ unsigned int PREC;
	int i = 3*(blockIdx.x*blockDim.x + threadIdx.x);
	int stride = blockDim.x * gridDim.x;
	PREC = 10000;
	while (i < N){
		unsigned char color = (prob[image[i+2]] * minmaxArray[8] + PREC * minmaxArray[6])/PREC;
		image[i+2] = color;
		i += stride;
	}
}

void CheckCudaError(char sms[], int line){
	hipError_t error;

	error = hipGetLastError();
	if(error){
		printf("(ERROR) %s - %s in %s at line %d\n", sms, hipGetErrorString(error), __FILE__, line);
		exit(EXIT_FAILURE);
	}
}

float GetTime(void){
	struct timeval tim;
	struct rusage ru;
	getrusage(RUSAGE_SELF, &ru);
	tim = ru.ru_utime;
	return ((double) tim.tv_sec + (double)tim.tv_usec / 1000000.0)*1000.0;
}

int minHisto(int i_histogram[], int l, int r){
	for(int i = l; i <= r; i++){
		if(i_histogram[i] != 0) return i;
	}
	return -1;
}

int maxHisto(int i_histogram[], int l, int r){
	for(int i = r; i >= l; i--){
		if(i_histogram[i] != 0) return i;
	}
	return -1;
}

int main(int argc, char** argv)
{
  unsigned int N;
  unsigned int numBytesImage, numBytesHisto, numBytesMinMax;
  unsigned int nBlocks, nThreads;
  int *d_histogramR, *d_histogramG, *d_histogramB, *d_minmaxArray;
  unsigned char *d_image;
  unsigned int *d_probR, *d_probG, *d_probB;
  float t1, t2, tiempoKHistoK, tiempoKEqualize, tiempoProcessing, tiempoHtoD, tiempoDtoH, tiempoKernels;

  hipEvent_t E1, E2, E3, E4, E5, E6;  

  // Ficheros de entrada y de salida 
  if (argc == 3) { fileIN = argv[1]; fileOUT = argv[2]; }
  else { printf("Usage: ./exe fileIN fileOUT (equalize)\n"); exit(0); }

  t1 = GetTime();
  printf("Reading image...\n");
  image = stbi_load(fileIN, &width, &height, &pixelWidth, 0);
  if (!image) {
    fprintf(stderr, "Couldn't load image.\n");
     return (-1);
  }
  printf("Image Read. Width : %d, Height : %d, nComp: %d\n",width,height,pixelWidth);

  for (int i = 0; i < 256; i++) {
	  histogramR[i] = 0; histogramG[i] = 0; histogramB[i] = 0;
  }
  printf("Filtrando\n");
  //Calculamos histograma:

  N = width * height * 3;
  nThreads = 256;
  nBlocks = (N + nThreads-1)/nThreads;

  numBytesImage = N * sizeof(unsigned char);
  numBytesHisto = 256 * sizeof(int);
  numBytesMinMax = 9 * sizeof(int);

  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);
  hipEventCreate(&E4);
  hipEventCreate(&E5);
  hipEventCreate(&E6);

  hipEventRecord(E5, 0);
  hipEventSynchronize(E5);
  hipMalloc((void**)&d_image, numBytesImage);
  hipMalloc((void**)&d_histogramR, numBytesHisto);
  hipMalloc((void**)&d_histogramG, numBytesHisto);
  hipMalloc((void**)&d_histogramB, numBytesHisto);
  hipMalloc((void**)&d_minmaxArray, numBytesMinMax);
  hipMalloc((void**)&d_probR, numBytesHisto);
  hipMalloc((void**)&d_probG, numBytesHisto);
  hipMalloc((void**)&d_probB, numBytesHisto);
  CheckCudaError((char*) "Error de Malloc ", __LINE__);
 
  hipMemcpy(d_image, image, numBytesImage, hipMemcpyHostToDevice);
  hipMemcpy(d_histogramR, histogramR, numBytesHisto, hipMemcpyHostToDevice);
  hipMemcpy(d_histogramG, histogramG, numBytesHisto, hipMemcpyHostToDevice);
  hipMemcpy(d_histogramB, histogramB, numBytesHisto, hipMemcpyHostToDevice);
  CheckCudaError((char*) "Error de Memcpy HtoD", __LINE__);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);
  HistoR<<<nBlocks, nThreads>>>(N, d_image, d_histogramR);
  HistoG<<<nBlocks, nThreads>>>(N, d_image, d_histogramG);
  HistoB<<<nBlocks, nThreads>>>(N, d_image, d_histogramB);
  CheckCudaError((char*) "Error de HistoK", __LINE__);
  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);
  
  hipMemcpy(histogramR, d_histogramR, numBytesHisto, hipMemcpyDeviceToHost);
  hipMemcpy(histogramG, d_histogramG, numBytesHisto, hipMemcpyDeviceToHost);
  hipMemcpy(histogramB, d_histogramB, numBytesHisto, hipMemcpyDeviceToHost);
  CheckCudaError((char*) "Error de Memcpy DtoH", __LINE__); 

  int minR = minHisto(histogramR, 0, 255), minG = minHisto(histogramG, 0, 255), minB = minHisto(histogramB, 0, 255);
  int maxR = maxHisto(histogramR, 0, 255), maxG = maxHisto(histogramG, 0, 255), maxB = maxHisto(histogramB, 0, 255);
  int maxminR = maxR - minR, maxminG = maxG - minG, maxminB = maxB - minB;
  unsigned int probR[256], probG[256], probB[256];
  int length = 256;
  probR[0] = PREC * histogramR[0]/(width*height);
  probG[0] = PREC * histogramG[0]/(width*height);
  probB[0] = PREC * histogramB[0]/(width*height);
  for(int i = 1; i < length; ++i){
	probR[i] = probR[i-1] + PREC * histogramR[i]/(width*height);
	probG[i] = probG[i-1] + PREC * histogramG[i]/(width*height);
	probB[i] = probB[i-1] + PREC * histogramB[i]/(width*height);
  }
  int auxMinmaxArray[9] = {minR, maxR, maxminR, minG, maxG, maxminG, minB, maxB, maxminB};

  hipMemcpy(d_minmaxArray, auxMinmaxArray, numBytesMinMax, hipMemcpyHostToDevice);
  hipMemcpy(d_probR, probR, numBytesHisto, hipMemcpyHostToDevice);
  hipMemcpy(d_probG, probG, numBytesHisto, hipMemcpyHostToDevice);
  hipMemcpy(d_probB, probB, numBytesHisto, hipMemcpyHostToDevice);
  CheckCudaError((char*) "Error de Memcpy HtoD", __LINE__);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);
  EqualizeR<<<nBlocks, nThreads>>>(N, d_image, d_minmaxArray, d_probR);
  EqualizeG<<<nBlocks, nThreads>>>(N, d_image, d_minmaxArray, d_probG);
  EqualizeB<<<nBlocks, nThreads>>>(N, d_image, d_minmaxArray, d_probB);
  CheckCudaError((char*) "Error de Equalize", __LINE__);
  hipEventRecord(E4, 0);
  hipEventSynchronize(E4);

  hipMemcpy(image, d_image, numBytesImage, hipMemcpyDeviceToHost);
  CheckCudaError((char*) "Error de Memcpy DtoH ", __LINE__);


  hipFree(d_image);
  hipFree(d_histogramR);
  hipFree(d_histogramG);
  hipFree(d_histogramB);
  hipFree(d_minmaxArray);
  hipFree(d_probR);
  hipFree(d_probG);
  hipFree(d_probB);
  CheckCudaError((char*) "Error de Free ", __LINE__);
  hipEventRecord(E6, 0);
  hipEventSynchronize(E6);

  hipEventElapsedTime(&tiempoKHistoK, E1, E2);
  hipEventElapsedTime(&tiempoKEqualize, E3, E4);
  hipEventElapsedTime(&tiempoProcessing, E5, E6);
  hipEventElapsedTime(&tiempoHtoD, E5, E1);
  hipEventElapsedTime(&tiempoDtoH, E4, E6);
  hipEventElapsedTime(&tiempoKernels, E1, E4);

  hipEventDestroy(E1);
  hipEventDestroy(E2);
  hipEventDestroy(E3);
  hipEventDestroy(E4);
  hipEventDestroy(E5);
  hipEventDestroy(E6);

  printf("Escribiendo\n");
  //ESCRITURA DE LA IMAGEN EN SECUENCIAL
  stbi_write_png(fileOUT,width,height,pixelWidth,image,0);
  t2 = GetTime();
  printf("---FILTRAR ColorSplitP---\n");
  printf("tiempo Global: %4.6f milseg\n", t2-t1);
  printf("tiempo HtoD: %4.6f milseg\n", tiempoHtoD);
  printf("tiempo DtoH: %4.6f milseg\n", tiempoDtoH);
  printf("tiempo Kernel HistoK: %4.6f milseg\n", tiempoKHistoK);
  printf("tiempo Kernel Equalize: %4.6f milseg\n", tiempoKEqualize);
  printf("tiempo Kernels juntos: %4.6f milseg\n", tiempoKernels);
  printf("tiempo Ecualización: %4.6f milseg\n", tiempoProcessing);
  printf("Ancho de banda: %4.2f MB/s\n", 0.000001*((width*height*3)/(tiempoProcessing*0.001)));
}

